#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <sys/time.h>

#define N 10000
#define Streams 10
#define DIMENSION 2
#define smooth  4
#define twoBB   2*smooth*smooth
float randn(float mu, float sigma);
float f_x(float x);

double timeDiff(struct timespec tStart, struct timespec tEnd){
   struct timespec diff;

   diff.tv_sec  = tEnd.tv_sec  - tStart.tv_sec  - (tEnd.tv_nsec<tStart.tv_nsec?1:0);
   diff.tv_nsec = tEnd.tv_nsec - tStart.tv_nsec + (tEnd.tv_nsec<tStart.tv_nsec?1000000000:0);

   return ((double) diff.tv_sec) + ((double) diff.tv_nsec)/1e9;
}

float randn(float mu, float sigma){
	float U1, U2, W, mult;
	static float X1, X2;
	static int call = 0;

	if (call == 1){
		call = !call;
		return (mu + sigma * (float)X2);
	}

	do{
		U1 = -1 + ((float)rand() / RAND_MAX) * 2;
		U2 = -1 + ((float)rand() / RAND_MAX) * 2;
		W = pow(U1, 2) + pow(U2, 2);
	} while (W >= 1 || W == 0);

	mult = sqrt((-2 * log(W)) / W);
	X1 = U1 * mult;
	X2 = U2 * mult;

	call = !call;

	return (mu + sigma * (float)X1);
}

float f_x(float x){
	return sin(0.02 * x) + sin(0.001 * x) + 0.1 * randn(0, 1);
}



__global__ void Kernel(float *x, float *y, float *expo, float *sumB, float *div)
{

    float twosmoothsqr = (float) 2*smooth*smooth;

    float soma1_16 = 0, soma2_16 = 0, soma3_16 = 0, soma4_16 = 0;
    float soma5_16 = 0,soma6_16 = 0,soma7_16 = 0, soma8_16 = 0;
    float soma9_16 = 0,soma10_16 = 0,soma11_16 = 0, soma12_16 = 0;
    float soma13_16 = 0,soma14_16 = 0,soma15_16 = 0, soma16_16 = 0;
    float soma = 0;
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int grid_width = gridDim.x * blockDim.x;

    int blockId = blockIdx.x + blockIdx.y  * gridDim.x; 
    int index = blockId * (blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;

    if(index_x < N && index_y < N){
        expo[index_y + index_x * N] = exp((-pow((x[index_x]-x[index_y]),2))/twosmoothsqr);
        expo[index_y + index_x * N + N*N] = exp((-pow((x[index_x]-x[index_y]),2))/twosmoothsqr)*y[index_y];
    }
    __syncthreads();

    /*if(index < 2*N){
        for(int i = 0; i< N ; i++){
            soma += expo[i + index*N];
        }
        sumB[index] = soma;
    }*/

    if(index < 2*N){
        for(int i = 0; i< N/16 ; i++){
            soma1_16 += expo[i + index*N];        
            soma2_16 += expo[i + N/16 + index*N];        
            soma3_16 += expo[i + 2*(N/16) + index*N];        
            soma4_16 += expo[i + 3*(N/16) + index*N];        
            soma5_16 += expo[i + 4*(N/16) + index*N];        
            soma6_16 += expo[i + 5*(N/16) + index*N];        
            soma7_16 += expo[i + 6*(N/16) + index*N];        
            soma8_16 += expo[i + 7*(N/16) + index*N];        
            soma9_16 += expo[i + 8*(N/16) + index*N];        
            soma10_16 += expo[i + 9*(N/16) + index*N];        
            soma11_16 += expo[i + 10*(N/16) + index*N];        
            soma12_16 += expo[i + 11*(N/16) + index*N];        
            soma13_16 += expo[i + 12*(N/16) + index*N];        
            soma14_16 += expo[i + 13*(N/16) + index*N];        
            soma15_16 += expo[i + 14*(N/16) + index*N];        
            soma16_16 += expo[i + 15*(N/16) + index*N];        
        }
        sumB[index] = soma1_16+soma2_16+soma3_16+soma4_16+soma5_16+soma6_16+soma7_16+soma8_16+soma9_16+soma10_16+soma11_16+soma12_16+soma13_16+soma14_16+soma15_16+soma16_16;
    }
    
    __syncthreads();

    if(index < N){
        div[index] = sumB[index + N] / sumB[index];
    }

}

int main()
{

	int device;
	int maxThreadsPerBlock;
	int blocksPerGrid;
	int Xsize, Ysize, exposize, sumBsize, Divsize;
    int sharedmem;
	float x[N], y[N], expo[N], Div[N];
	float *d_X = NULL ,*d_Y = NULL,*d_expo = NULL,*d_sumA = NULL, *d_sumB = NULL,*d_Div = NULL;
    FILE *fp;
    struct timespec timeVect[20];
    double timeCPU, timeGPU[20];
    hipError_t err[] = { hipSuccess , hipSuccess , hipSuccess };
    hipError_t mem = hipSuccess;
    if (err[0] != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err[0]));
        exit(EXIT_FAILURE);
    }
    clock_gettime(CLOCK_REALTIME, &timeVect[9]);
    hipGetDevice(&device);
    clock_gettime(CLOCK_REALTIME, &timeVect[10]);
    hipFree(0);

	   float sumA, sumB, yest_cpu[N];
    
    // Allocate the host
    float *h_X = (float *)malloc(N * sizeof(float));
    float *h_Y = (float *)malloc(N * sizeof(float));
    float *yest = (float *)malloc(N * sizeof(float));

    // Verify that allocations succeeded
    if (h_X == NULL || h_Y == NULL || yest == NULL )
    {
        fprintf(stderr, "Failed to allocate host data!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input data
    for (int i = 0; i < N ; i++ ){
        h_X[i] = (i*1.0)/10;
        h_Y[i] = f_x(h_X[i]);
        yest[i] = 1;
    }

    // Compute expected result
    printf("Performing the computation on the CPU...\n");
    clock_gettime(CLOCK_REALTIME, &timeVect[0]);
    for(int i=0;i<N;i++){
        sumA=0;
        sumB=0;
        for(int j=0;j<N;j++){
            sumA = sumA + exp((-pow((h_X[i]-h_X[j]),2))/(2*pow(smooth,2)))*h_Y[j];
            sumB = sumB + exp((-pow((h_X[i]-h_X[j]),2))/(2*pow(smooth,2)));
        }
        yest_cpu[i] = sumA / sumB;
    }
    clock_gettime(CLOCK_REALTIME, &timeVect[1]);
    timeCPU = timeDiff(timeVect[0],timeVect[1]);
	Xsize = N * sizeof(float);
	Ysize = N * sizeof(float);
    exposize = N * sizeof(float);
    sumBsize = N * sizeof(float);
    Divsize = N * sizeof(float);
	


    struct hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device);
    
    size_t uCurAvailMemoryInBytes;
    size_t uTotalMemoryInBytes;
    hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
    int mem_av = uTotalMemoryInBytes;
	printf("Device Number: %d\n", device);
	printf("\tDevice name: %s\n", props.name);
	printf("\tDevice max threads per block: %d\n", props.maxThreadsPerBlock);
	printf("\tMemory Clock Rate (KHz): %d\n", props.memoryClockRate);
	printf("\tMemory Bus Width (bits): %d\n",props.memoryBusWidth);
    printf("\tTotal Memory Available (MB): %d\n",uTotalMemoryInBytes / ( 1024 * 1024 ));
    printf("\tShared memory: %d\n",props.sharedMemPerBlock);

    sharedmem = props.sharedMemPerBlock;
	maxThreadsPerBlock = props.maxThreadsPerBlock;
    // create 2d 2x2 thread block
    dim3 block_size;
    block_size.x = sqrt(maxThreadsPerBlock);
    block_size.y = sqrt(maxThreadsPerBlock);
    printf("Creating thread block of %dx%d threads...\n",block_size.x,block_size.y);
    //configure 2d grid
    dim3 grid_size;
    grid_size.x = sqrt((N*N) / maxThreadsPerBlock) + 1;
    grid_size.y = sqrt((N*N) / maxThreadsPerBlock) + 1;
    printf("Creating grid of %dx%d blocks...\n",grid_size.x,grid_size.y);
    clock_gettime(CLOCK_REALTIME, &timeVect[0]);
    err[0] = hipMalloc((void**)&d_X, Xsize);
    if (err[0] != hipSuccess) {
        fprintf(stderr, "hipMalloc failed X!");
        return 0;
    }
    err[0] = hipMalloc((void**)&d_Y, Ysize);
    if (err[0] != hipSuccess) {
        fprintf(stderr, "hipMalloc failed Y!");
        return 0;
    }
    err[0] = hipMalloc((void**)&d_expo,exposize*N*2);
    if (err[0] != hipSuccess) {
        fprintf(stderr, "hipMalloc failed expo!");
        return 0;
    }

    err[0] = hipMalloc((void**)&d_sumB,sumBsize*2);
    if (err[0] != hipSuccess) {
        fprintf(stderr, "hipMalloc failed sumB!");
        return 0;
    }
    err[0] = hipMalloc((void**)&d_Div,Divsize);
    if (err[0] != hipSuccess) {
        fprintf(stderr, "hipMalloc failed sumB!");
        return 0;
    }
    clock_gettime(CLOCK_REALTIME, &timeVect[1]);
    hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
    int mem_curr = uCurAvailMemoryInBytes;
    int mem_all = (mem_av - mem_curr)/(1024*1024);
    printf("Allocating %d MB of memory...\n",mem_all);

    printf("Copying from CPU to GPU...\n");
    err[0] = hipMemcpy(d_X, h_X, Xsize , hipMemcpyHostToDevice);
    err[1] = hipMemcpy(d_Y, h_Y, Ysize , hipMemcpyHostToDevice);

    clock_gettime(CLOCK_REALTIME, &timeVect[2]);
    if ((err[0] != hipSuccess) || (err[1] != hipSuccess) || (err[2] != hipSuccess)){
        fprintf(stderr, "Failed to allocate device values X or Y! Error codes are:\n");
        fprintf(stderr, "\t Allocation of %d Bytes for value X: %s\n", Xsize , hipGetErrorString(err[0]) );
        fprintf(stderr, "\t Allocation of %d Bytes for value Y: %s\n", Ysize , hipGetErrorString(err[1]) );
        fprintf(stderr, "\t Allocation of %d Bytes for value Yest: %s\n", Ysize*N , hipGetErrorString(err[2]) );
        exit(EXIT_FAILURE);
    }
    
    clock_gettime(CLOCK_REALTIME, &timeVect[3]);
    Kernel<<<grid_size, block_size>>>(d_X,d_Y,d_expo,d_sumB,d_Div);
    clock_gettime(CLOCK_REALTIME, &timeVect[4]);

    float somaA[N], somaB[2*N];

    /*err[0] = hipMemcpy(somaA, d_sumA, N*sizeof(float) , hipMemcpyDeviceToHost);
    if (err[0] != hipSuccess){
        fprintf(stderr, "Failed to copy Yest from device to host (error code %s)!\n", hipGetErrorString(err[0]));
         exit(EXIT_FAILURE);*/
    clock_gettime(CLOCK_REALTIME, &timeVect[5]);
    err[0] = hipMemcpy(Div, d_Div, N*sizeof(float), hipMemcpyDeviceToHost);
    clock_gettime(CLOCK_REALTIME, &timeVect[6]);
    if (err[0] != hipSuccess){
        fprintf(stderr, "Failed to copy Yest from device to host (error code %s)!\n", hipGetErrorString(err[0]));
         exit(EXIT_FAILURE);
    }
    fp=fopen("output.out", "w");
    for(int j=0;j<N;j++){
        fprintf(fp,"%f\n",Div[j]);
    }
    fclose(fp);

    clock_gettime(CLOCK_REALTIME, &timeVect[7]);
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_expo);
    hipFree(d_sumB);
    hipFree(d_Div);
    clock_gettime(CLOCK_REALTIME, &timeVect[8]);

    timeGPU[0] = timeDiff(timeVect[0],timeVect[1]);
    timeGPU[1] = timeDiff(timeVect[1],timeVect[2]);
    timeGPU[2] = timeDiff(timeVect[3],timeVect[4]);
    timeGPU[3] = timeDiff(timeVect[5],timeVect[6]);
    timeGPU[4] = timeDiff(timeVect[7],timeVect[8]);
    timeGPU[5] = timeDiff(timeVect[0],timeVect[8]);
    timeGPU[6] = timeDiff(timeVect[9],timeVect[10]);

    printf("    ... execution took %.6f seconds (speedup=%.3f), corresponding to:\n",timeGPU[5]+timeGPU[6],timeCPU/(timeGPU[5]+timeGPU[6]));
    printf("          - first call to the device           -> %.6f seconds\n",timeGPU[6]);
    printf("          - allocation of memory on the device -> %.6f seconds\n",timeGPU[0]);
    printf("          - copying data from host to device   -> %.6f seconds\n",timeGPU[1]);
    printf("          - kernel execution on the device     -> %.6f seconds\n",timeGPU[2]);
    printf("          - copying data from device to host   -> %.6f seconds\n",timeGPU[3]);
    printf("          - freeing data on the device         -> %.6f seconds\n",timeGPU[4]);
    printf("----------------------------------------------------------------------------\n");
    
    int i=0,j=0;

    for (i = 0, j = 0; i < N; i++)
    {
        if (fabs(Div[i]-yest_cpu[i]) > 1e-3)
        {
            //fprintf(stderr, "Result verification failed at element %d => CPU returns %f while GPU returns %f\n", i, yest_cpu[i],Div[i]);
            j++;
        }
    }
    if (j>0) {
        printf("%d errors found!\n",j);
        exit(EXIT_FAILURE);
    }
    printf("Test PASSED\n");

    printf("Done\n");


    return 0;

}
