#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#define N		10000
#define smooth	4
#define THREADS_PER_BLOCK 1000

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 */
__global__ void calcy(float *X, float *Y, float *Yest,int indice) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j;
	
	float smoothing = (float) smooth;
	float A=0, B=0, tmp;
	float smth = 2*pow(smoothing,2);
	
	for(j=0;j<N;j++){
		tmp = exp((-pow((X[i+indice*N]-X[j+indice*N]),2))/(smth));
		A = A + tmp*Y[j+indice*N];
		B = B + tmp;
	}
	
	Yest[i] = A/B;
}

/**
 * timeDiff
 *
 * Computes the difference (in ns) between the start and end time
 */
double timeDiff(struct timespec tStart, struct timespec tEnd){
   struct timespec diff;

   diff.tv_sec  = tEnd.tv_sec  - tStart.tv_sec  - (tEnd.tv_nsec<tStart.tv_nsec?1:0);
   diff.tv_nsec = tEnd.tv_nsec - tStart.tv_nsec + (tEnd.tv_nsec<tStart.tv_nsec?1000000000:0);

   return ((double) diff.tv_sec) + ((double) diff.tv_nsec)/1e9;
}

/**
 * randn
 * 
 * Computes a random value with a gaussian distribuition
 */
double randn (double mu, double sigma){
    double U1, U2, W, mult;
    static double X1, X2;
    static int call = 0;
    
    if (call == 1){
        call = !call;
        return (mu + sigma * (double) X2);
    }
    
    do{
        U1 = -1 + ((double) rand () / RAND_MAX) * 2;
        U2 = -1 + ((double) rand () / RAND_MAX) * 2;
        W = pow (U1, 2) + pow (U2, 2);
    }while (W >= 1 || W == 0);
    
    mult = sqrt ((-2 * log (W)) / W);
    X1 = U1 * mult;
    X2 = U2 * mult;
    
    call = !call;
    
    return (mu + sigma * (double) X1);
}

/**
 * f_x
 * 
 * Computes y = f(x)
 */
float f_x(float x){
	return sin(0.02 * x) + sin(0.001 * x) + 0.1 * randn(0, 1);
}

/**
 * Host main routine
 */
int main(int argc, char **argv) {
    // Error code to check return values for CUDA calls
    unsigned i,j;
    struct timespec timeVect[7];
    double timeCPU, timeGPU[7];
    hipError_t err[] = { hipSuccess , hipSuccess , hipSuccess };
	if(argc < 2){
		printf("N indefinido\n");
		exit(-1);
	}
	int MAX = atoi(argv[1]);
    //cpu variables
	float sumA, sumB, yest_cpu[MAX];
	FILE* fp;
	
	for(int i = 0; i <7; i++)
		timeGPU[i] = 0;

    // Allocate the host
    float *h_X = (float *)malloc(MAX * sizeof(float));
    float *h_Y = (float *)malloc(MAX * sizeof(float));
    float *yest = (float *)malloc(MAX * sizeof(float));

    // Verify that allocations succeeded
    if (h_X == NULL || h_Y == NULL || yest == NULL )
    {
        fprintf(stderr, "Failed to allocate host data!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input data
    for (i = 0; i < MAX ; i++ ){
		h_X[i] = (i*1.0)/10;
		h_Y[i] = f_x(h_X[i]);
		yest[i] = 0;
	}

    // Compute expected result
    printf("Performing the computation on the CPU...\n");
	clock_gettime(CLOCK_REALTIME, &timeVect[0]);
    for(i=0;i<MAX;i++){
		sumA=0;
		sumB=0;
		for(j=0;j<MAX;j++){
			sumA = sumA + exp((-pow((h_X[i]-h_X[j]),2))/(2*pow(smooth,2)))*h_Y[j];
		}
		for(j=0;j<MAX;j++){
			sumB = sumB + exp((-pow((h_X[i]-h_X[j]),2))/(2*pow(smooth,2)));
		}
		yest_cpu[i] = sumA / sumB;
	}
    clock_gettime(CLOCK_REALTIME, &timeVect[1]);
    timeCPU = timeDiff(timeVect[0],timeVect[1]);
    printf("    ... execution took %.6f seconds\n", timeCPU );

    // Compute on the GPU
    printf("----------------------------------------------------------------------------\n");
    printf("Performing the computation on the GPU...\n");

    // initialize the device (just measure the time for the first call to the device)
	//hipSetDevice(0);
	//hipDeviceReset();
    clock_gettime(CLOCK_REALTIME, &timeVect[0]);
    hipFree(0);
    clock_gettime(CLOCK_REALTIME, &timeVect[1]);

    // Allocate memory on the device
    printf(" ... Allocation of memory on the Device ...\n");
    float *d_X = NULL , *d_Y = NULL , *d_yest = NULL;
    err[0] = hipMalloc( (void **) &d_X , MAX * sizeof(float) );
    err[1] = hipMalloc( (void **) &d_Y , MAX * sizeof(float) );
    err[2] = hipMalloc( (void **) &d_yest , N * sizeof(float) );

    if ((err[0] != hipSuccess) || (err[1] != hipSuccess) || (err[2] != hipSuccess))
    {
        fprintf(stderr, "Failed to allocate device memory! Error codes are:\n");
        fprintf(stderr, "\t Allocation of %d Bytes for X: %s\n", N * sizeof(float) , hipGetErrorString(err[0]) );
        fprintf(stderr, "\t Allocation of %d Bytes for Y: %s\n", N * sizeof(float) , hipGetErrorString(err[1]) );
        fprintf(stderr, "\t Allocation of %d Bytes for yest: %s\n", N * sizeof(float) , hipGetErrorString(err[2]) );
        exit(EXIT_FAILURE);
    }


    // Copy the host input data to the device memory
	clock_gettime(CLOCK_REALTIME, &timeVect[2]);
    printf(" ... Copying input data from the host memory to the CUDA device ...\n");
    err[0] = hipMemcpy(d_X, h_X, MAX * sizeof(float) , hipMemcpyHostToDevice);
    err[1] = hipMemcpy(d_Y, h_Y, MAX * sizeof(float) , hipMemcpyHostToDevice);
    clock_gettime(CLOCK_REALTIME, &timeVect[3]);
    timeGPU[2] = timeDiff(timeVect[2],timeVect[3]);

    if ((err[0] != hipSuccess) || (err[1] != hipSuccess))
    {
        fprintf(stderr, "Failed to copy data to the device! Error codes are:\n");
        fprintf(stderr, "\t X: %s\n", hipGetErrorString(err[0]) );
        fprintf(stderr, "\t Y: %s\n", hipGetErrorString(err[1]) );
        exit(EXIT_FAILURE);
    }


    // Launch the CUDA Kernel

    dim3 tpb (THREADS_PER_BLOCK);
	dim3 bpg (N/THREADS_PER_BLOCK);
	
	
    printf(" ... CUDA kernel launch ...\n");
	for(int i = 0 ; i < ceil(MAX/N) ; i++){
		clock_gettime(CLOCK_REALTIME, &timeVect[3]);
		calcy<<<bpg, tpb>>>(d_X, d_Y, d_yest,i);
		clock_gettime(CLOCK_REALTIME, &timeVect[4]);
		err[0] = hipGetLastError();

		if (err[0] != hipSuccess)
		{
			fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err[0]));
			exit(EXIT_FAILURE);
		}
		timeGPU[3] = timeGPU[3] + timeDiff(timeVect[3],timeVect[4]);

		// Copy the result back to host memory
		clock_gettime(CLOCK_REALTIME, &timeVect[4]);
		err[0] = hipMemcpy(yest+i*N, d_yest, N * sizeof(float) , hipMemcpyDeviceToHost);
		clock_gettime(CLOCK_REALTIME, &timeVect[5]);
		timeGPU[4] = timeGPU[4] + timeDiff(timeVect[4],timeVect[5]);
		printf("Copy output data from the CUDA device to the host memory in %.6f seconds\n",timeDiff(timeVect[4],timeVect[5]));
		if (err[0] != hipSuccess)
		{
			fprintf(stderr, "Failed to copy result from device to host (error code %s)!\n", hipGetErrorString(err[0]));
			exit(EXIT_FAILURE);
		}

	}
	clock_gettime(CLOCK_REALTIME, &timeVect[5]);
    err[0] = hipFree(d_X);
    err[1] = hipFree(d_Y);
    err[2] = hipFree(d_yest);
	clock_gettime(CLOCK_REALTIME, &timeVect[6]);

    if ((err[0] != hipSuccess) || (err[1] != hipSuccess) || (err[2] != hipSuccess))
    {
        fprintf(stderr, "Failed to free device memory!\n");
        fprintf(stderr, "\t X: %s\n", hipGetErrorString(err[0]) );
        fprintf(stderr, "\t Y: %s\n", hipGetErrorString(err[1]) );
        fprintf(stderr, "\t d_yest: %s\n", hipGetErrorString(err[2]) );
        exit(EXIT_FAILURE);
    }
    clock_gettime(CLOCK_REALTIME, &timeVect[6]);
    timeGPU[0] = timeDiff(timeVect[0],timeVect[1]);
    timeGPU[1] = timeDiff(timeVect[1],timeVect[2]);
    //timeGPU[3] = timeDiff(timeVect[3],timeVect[4]);
    //timeGPU[4] = timeDiff(timeVect[4],timeVect[5]);
    timeGPU[5] = timeDiff(timeVect[5],timeVect[6]);
    timeGPU[6] = timeGPU[1] + timeGPU[2] + timeGPU[3] + timeGPU[4];
    printf("    ... execution took %.6f seconds, corresponding to:\n",timeGPU[6]);
    printf("          - first call to the device           -> %.6f seconds\n",timeGPU[0]);
    printf("          - allocation of memory on the device -> %.6f seconds\n",timeGPU[1]);
    printf("          - copying data from host to device   -> %.6f seconds\n",timeGPU[2]);
    printf("          - kernel execution on the device     -> %.6f seconds\n",timeGPU[3]);
    printf("          - copying data from device to host   -> %.6f seconds\n",timeGPU[4]);
    printf("          - freeing data on the device         -> %.6f seconds\n",timeGPU[5]);
    printf("----------------------------------------------------------------------------\n");
	printf("    ... overall speedup=%.3f and kernel only execution speedup=%.3f\n",timeCPU/timeGPU[6], timeCPU/timeGPU[3]);
	printf("----------------------------------------------------------------------------\n");
	
    //write data to file
	fp=fopen("enunciado.txt", "a");
	fwrite("X,Y,CPU,GPU\n",12,1,fp);
	fprintf(fp,"%.6f,%.6f,%.6f,%.6f,%.6f,%.6f,%.6f,%.6f\n",timeCPU, timeGPU[6], timeGPU[0],timeGPU[1], timeGPU[2],timeGPU[3], timeGPU[4],timeGPU[5]);
	fclose(fp);
	
	// Free host memory
    //free(h_X);
    //free(h_Y);
    //free(yest);

    // Reset the device and exit
    err[0] = hipDeviceReset();

    if (err[0] != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err[0]));
        exit(EXIT_FAILURE);
    }
	
	// Verify that the result matrix is correct
    for (i = 0, j = 0; i < MAX; i++)
    {
        if (fabs(yest[i]-yest_cpu[i]) > 1e-6)
        {
            //fprintf(stderr, "Result verification failed at element %d => CPU returns %f while GPU returns %f\n", i, yest_cpu[i],yest[i]);
            j++;
        }
    }
    float erro = j/MAX;
    if (j>0) {
        printf("%d errors found! --- %f of %d Elements\n",j,erro,MAX);
        exit(EXIT_FAILURE);
    }
    printf("Test PASSED\n");

    printf("Done\n");
    return 0;
}
